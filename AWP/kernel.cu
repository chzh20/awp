#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <cstdio>
#include <map>
#include <algorithm>
#include <iostream>
#include <vector>
#include <thrust/sort.h> 
#include "Model.h"
#include "Timer.h"
#include "Point.h"
#include "Model_vector.h"
#include "AWP.h"

using namespace std;

#define TID threadIdx.x
#define BID blockIdx.x
#define BMX blockDim.x
#define SIZE_WINDOWS_HE 40//4 
#define huatu 0
#define INF 9999.0
#define TEST_AVG 1
#define EPCH_AVG 1
#define DEBUG 0
#define WU 0
#define TEST_MAX_MIN 0
#define TEST_ZT 1
#define CUT 1
#define SSD 0 
#define MSD 1
/*#define EB 16
#define VB 16
#define EE 13063
#define VV 2178
*/
typedef char uchar;
/*source point: 512
torus.ply 60 4
bunny.obj 512 50 282 0.558262
golf: 1 45 430
dragon: 512 403 40
*/
//Model model = Model("model/bunny_nf144k.m");

int sid = -1;
char outPath[256] = { '\0' };
char fileName[256] = { '\0' };

void dealInput(int argc, char *argv[]) {
	for (int i = 1; i < argc;) {
		if (strcmp(argv[i], "-s") == 0) {
			sid = atoi(argv[i + 1]);
			i += 2;
		}
		else if (strcmp(argv[i], "-o") == 0) {
			strcpy(outPath, argv[i + 1]);
			i += 2;
		}
		else if (strcmp(argv[i], "-m") == 0) {
			strcpy(fileName, argv[i + 1]);
			i += 2;
		}
		else i++;
	}
}

int main(int argc, char* argv[]) {
	dealInput(argc, argv);
	if (sid < 0) {
		puts("No source!");
		return 0;
	}
	if (strlen(fileName) == 0) {
		puts("No model!");
		return 0;
	}
	AWP *awp = new AWP(fileName); 
	//model = Model_vector(argv[1]);
	cout << "------------------------load model begin------------------------\n";
	cout << "File name:\t" << awp->model.GetFileName() << endl;
	try {
		awp->model.LoadModel();
	}
	catch (const char* msg) {
		cout << "ERRORS happen!\n" << msg << endl;
		return 1;
	}
	awp->model.dist = new float[awp->model.GetNumOfVerts()];
	Point3D maxP = awp->model.vertexs[0], minP = awp->model.vertexs[1];
	for (int i = 0; i < awp->model.GetNumOfVerts(); i++) {
		maxP.x = max(maxP.x, awp->model.vertexs[i].x);
		maxP.y = max(maxP.y, awp->model.vertexs[i].y);
		maxP.z = max(maxP.z, awp->model.vertexs[i].z);
		minP.x = min(minP.x, awp->model.vertexs[i].x);
		minP.y = min(minP.y, awp->model.vertexs[i].y);
		minP.z = min(minP.z, awp->model.vertexs[i].z);
	}
	float len = getDistance(maxP, minP) / sqrt(3.0);
	for (int i = 0; i < awp->model.GetNumOfVerts(); i++) {
		awp->model.vertexs[i].x = (awp->model.vertexs[i].x - minP.x) / len;
		awp->model.vertexs[i].y = (awp->model.vertexs[i].y - minP.y) / len;
		awp->model.vertexs[i].z = (awp->model.vertexs[i].z - minP.z) / len;
	}
	awp->model.creatHalfEdge();
	try {
		awp->model.findAllInfoOnVers();
	}
	catch (const char* msg) {
		cout << "ERRORS happen!\n" << endl;
		return 1;
	}
	if (!awp->model.isClose) {
		cout << "It is an open model!\n";
		return 0;
	}
	else {
		cout << "It is a close model\n";
	}
	
	//system("pause"); 
	printf("The number of face:\t%d\n", awp->model.GetNumOfFaces());
	printf("The number of vertex:\t%d\n", awp->model.GetNumOfVerts());
	printf("The number of edge:\t%d\n", awp->model.GetNumOfEdges()); 
	cout << "-------------------------load model end-------------------------\n\n";
	//sid = getnum(argv[2]);
	do { 
		awp->run(sid);
		puts("------------------------Output the result-----------------------");
		puts("Outputing.....");
		strcpy(awp->outPath, outPath);
		awp->outVt();
		awp->outInfo();
		puts("----------------------------The end!----------------------------");

		//break;
		puts("If you want to continue use this model, you can input new id of start vertex and new output mesh file. Such as 512 512_outInfo.obj\n Else if you input -1 to end.");
		scanf("%d", &sid); 
		if (sid < 0 || sid >= awp->model.GetNumOfVerts()) break;
		scanf("%s", outPath);
	} while (sid != -1);
	return 0;
}
/*
0 0 0
0 0 1
0 1 0
1 0 0
1 1 1
2 1 0
2 0 3
0 1 3
*/